#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include "utilities.h"

__global__ void stochasticTensorGenerator(const float* inputData, const float* randomMatrix, int8_t* output, int inputData_size, int RM_cols);

#define cudaCheckError(err) (cudaCheck(err, __FILE__, __LINE__))
inline void cudaCheck(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error at " << file << ":" << line << " - " 
                  << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Helper function to flatten a 2D vector into a 1D vector
std::vector<float> flatten2D(const std::vector<std::vector<double>>& input) {
    std::vector<float> output;
    for (const auto& vec : input) {
        for (const auto& val : vec) {
            output.push_back(static_cast<float>(val));
        }
    }
    return output;
}

template <typename T>
std::vector<float> flatten(const std::vector<std::vector<T>>& input) {
    std::vector<float> output;
    for (const auto& vec : input) {
        for (const auto& val : vec) {
            output.push_back(static_cast<float>(val));
        }
    }
    return output;
}

// stochasticTensorGenerator
__global__ void stochasticTensorGenerator(const float* inputData, const float* randomMatrix, int8_t* output, int inputData_size, int RM_cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < inputData_size * RM_cols) {
        int n_idx = idx / RM_cols;
        int rm_col_idx = idx % RM_cols;
        int rm_idx = n_idx * RM_cols + rm_col_idx;
        
        // Determine if inputData[n_idx] is positive or negative
        if (inputData[n_idx] > 0) {
            // Add 1 to all positions separated by RM_cols
            if (rm_col_idx == 0) {
                output[idx] = 1;
            } else {
                output[idx] = (randomMatrix[rm_idx] < (inputData[n_idx])) ? 1 : 0;
            }
        } else if (inputData[n_idx] < 0) {
            // Add -1 to all positions separated by RM_cols
            if (rm_col_idx == 0) {
                output[idx] = -1;
            } else {
                output[idx] = (randomMatrix[rm_idx] < ((inputData[n_idx] * -1))) ? 1 : 0;
            }
        } else if (inputData[n_idx] == 0) {
            output[idx] = 0;
        } 
    }
}

__constant__ int inputWidthConv2;
__constant__ int KernelHeightConv2;
__constant__ int KernelWidthConv2;
__constant__ int OutputHeightConv2;
__constant__ int OutputWidthConv2;
__constant__ int NConv2;

// CUDA kernel for 2D convolution
__global__ void conv2D(const int8_t* input, const int8_t* kernel, float* output) {
    //extern __shared__ int shared_result[];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < OutputHeightConv2 && j < OutputWidthConv2) {
        float accumulatedOnes_pos = 0;
        float accumulatedOnes_neg = 0;
        int numberOfAccumulations = 0;
        for (size_t m = 0; m < KernelHeightConv2; ++m) {
            for (size_t n = 0; n < KernelWidthConv2; ++n) {
                numberOfAccumulations++;
                //count = 0;
                if((input[((i + m) * inputWidthConv2 * NConv2 + (j + n) * NConv2)]) * (kernel[(m * KernelWidthConv2 * NConv2 + n * NConv2)]) == 1){
                    for (size_t bit_counter = 1; bit_counter < NConv2; ++bit_counter) {
                        if(((input[((i + m) * inputWidthConv2 * NConv2 + (j + n) * NConv2)+bit_counter] & kernel[(m * KernelWidthConv2 * NConv2 + n * NConv2)+bit_counter])) == 1)
                        {accumulatedOnes_pos++;}
                    }
                } else if ((input[((i + m) * inputWidthConv2 * NConv2 + (j + n) * NConv2)]) * (kernel[(m * KernelWidthConv2 * NConv2 + n * NConv2)])== -1){
                    for (size_t bit_counter = 1; bit_counter < NConv2; ++bit_counter) {
                        if(((input[((i + m) * inputWidthConv2 * NConv2 + (j + n) * NConv2)+bit_counter] & kernel[(m * KernelWidthConv2 * NConv2 + n * NConv2)+bit_counter])) == 1)
                        {accumulatedOnes_neg++;}
                    }
                } 
            }
        }
        output[i * OutputWidthConv2 + j] = accumulatedOnes_pos - accumulatedOnes_neg;
    }
}

std::vector<std::vector<float>> ScCudaConv2d(
    const std::vector<float>& polarInputData, 
    const std::vector<float>& polarKernelData, 
    const std::vector<float>& randomMatrix_Input, 
    const std::vector<float>& randomMatrix_Kernel,
    const int bitstream_Length,
    const int height,
    const int width,
    const int heightK){

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    // Determine the type of the input vector (2D or 3D)
    std::vector<float> inputData = polarInputData;
    std::vector<float> RM_flat = randomMatrix_Input;

    int inputData_size = inputData.size();
    int RM_cols = bitstream_Length;
    int output_size = inputData_size * RM_cols;

    float* d_inputData;
    float* d_RM;
    int8_t* d_output;

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (output_size + blockSize - 1) / blockSize;

    // Allocate input device memory
    cudaCheckError(hipMalloc(&d_inputData, inputData_size * sizeof(float)));
    cudaCheckError(hipMalloc(&d_RM, inputData_size * RM_cols * sizeof(float)));
    cudaCheckError(hipMalloc(&d_output, output_size * sizeof(int8_t)));

    ///SEPARATION KERNEL VARIABLES
    std::vector<float> inputDataK = polarKernelData;
    std::vector<float> RM_flatK = randomMatrix_Kernel;

    int inputData_sizeK = inputDataK.size();
    int RM_colsK = bitstream_Length;
    int output_sizeK = inputData_sizeK * RM_colsK;

    float* d_inputDataK;
    float* d_RMK;
    int8_t* d_outputK;

    // Define grid kernel and block kernel dimensions
    int blockSizeK = 256;
    int numBlocksK = (output_sizeK + blockSizeK - 1) / blockSizeK;      

    // Allocate kernel device memory
    cudaCheckError(hipMalloc(&d_inputDataK, inputData_sizeK * sizeof(float)));
    cudaCheckError(hipMalloc(&d_RMK, inputData_sizeK * RM_colsK * sizeof(float)));
    cudaCheckError(hipMalloc(&d_outputK, output_sizeK * sizeof(int8_t)));

    /////////////////////////////// SC CONV2D KERNEL ///////////////////////////////////////

    const int N = RM_cols; // bitstream length

    int inputHeight = height, inputWidth = width;
    int kernelHeight = heightK, kernelWidth = heightK;
    int outputHeight = inputHeight - kernelHeight + 1;
    int outputWidth = inputWidth - kernelWidth + 1;

    // std::cout << "inputHeight: " << inputHeight << std::endl; 
    // std::cout << "inputWidth: " << inputWidth << std::endl; 
    // std::cout << "kernelHeight: " << kernelHeight << std::endl;
    // std::cout << "outputHeight: " << outputHeight << std::endl; 
    // std::cout << "outputWidth: " << outputWidth << std::endl;

    float* h_output = new float[outputHeight * outputWidth];

    float* d_outputConv2;

                    //time measure
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);

    hipMemcpyToSymbol(HIP_SYMBOL(inputWidthConv2), &inputWidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(KernelHeightConv2), &kernelHeight, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(KernelWidthConv2), &kernelWidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(OutputHeightConv2), &outputHeight, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(OutputWidthConv2), &outputWidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(NConv2), &N, sizeof(int));

    cudaCheckError(hipMalloc(&d_outputConv2, outputHeight * outputWidth * sizeof(float)));

    // Define grid and block dimensions
    dim3 blockDim(32, 32);
    dim3 gridDim((outputWidth + blockDim.x - 1) / blockDim.x, (outputHeight + blockDim.y - 1) / blockDim.y);

    // Copy INPUT data to device
    cudaCheckError(hipMemcpyAsync(d_inputData, inputData.data(), inputData_size * sizeof(float), hipMemcpyHostToDevice, stream1));
    cudaCheckError(hipMemcpyAsync(d_RM, RM_flat.data(), inputData_size * RM_cols * sizeof(float), hipMemcpyHostToDevice, stream1));

    // Copy KERNEL data to device
    cudaCheckError(hipMemcpyAsync(d_inputDataK, inputDataK.data(), inputData_sizeK * sizeof(float), hipMemcpyHostToDevice, stream2));
    cudaCheckError(hipMemcpyAsync(d_RMK, RM_flatK.data(), inputData_sizeK * RM_colsK * sizeof(float), hipMemcpyHostToDevice, stream2));

    // Launch input_data kernel
    stochasticTensorGenerator<<<numBlocks, blockSize, 0, stream1>>>(d_inputData, d_RM, d_output, inputData_size, RM_cols);
    cudaCheckError(hipGetLastError());
    
    // Launch kernel_data kernel
    stochasticTensorGenerator<<<numBlocksK, blockSizeK, 0, stream2>>>(d_inputDataK, d_RMK, d_outputK, inputData_sizeK, RM_colsK);
    cudaCheckError(hipGetLastError());

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Launch the convolution kernel
    conv2D<<<gridDim, blockDim>>>(d_output, d_outputK, d_outputConv2);
    cudaCheckError(hipGetLastError());

    //hipDeviceSynchronize();

    // Copy result back to host
    cudaCheckError(hipMemcpy(h_output, d_outputConv2, outputHeight * outputWidth * sizeof(float), hipMemcpyDeviceToHost));

    std::vector<std::vector<float>> outputConv2(outputHeight, std::vector<float>(outputWidth));

                //STOP time measure
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "\n" << std::endl;
            std::cout << "C++ CONV2D End-to-end execution time: " << milliseconds << " ms_" << std::endl;   

    for (int i = 0; i < outputHeight; ++i) {
        for (int j = 0; j < outputWidth; ++j) {
            //outputConv2[i][j] = static_cast<double>(h_output[i * outputWidth + j]);
            outputConv2[i][j] = h_output[i * outputWidth + j];
            //std::cout << "outputConv2[i][j] " << outputConv2[i][j] << std::endl;
        }
    }


    // Free device memory
    hipFree(d_inputData);
    hipFree(d_RM);
    hipFree(d_output);

    hipFree(d_inputDataK);
    hipFree(d_RMK);
    hipFree(d_outputK);

    hipFree(d_outputConv2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Clean up
    delete[] h_output;

    return outputConv2;
}

__constant__ int input_sizeFcL;
__constant__ int output_sizeFcL;
__constant__ int bit_lengthFcL;

///////////////////CUDA FULLY CONNECTED LAYER/////////////////////////
// Kernel function for forward pass of a fully connected layer
__global__ void forward_pass(
    int8_t* input, int8_t* weights, int8_t* biases, float* output) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < output_sizeFcL) {
        float accumulatedOnes_pos= 0;
        float accumulatedOnes_neg = 0;
        int numberOfAccumulations = 0;

        for (size_t i = 0; i < input_sizeFcL; ++i) {
            numberOfAccumulations++;

            if((input[(i * bit_lengthFcL)]) * (weights[((i * output_sizeFcL + j) * bit_lengthFcL)]) == 1){
                for (size_t k = 1; k < bit_lengthFcL; ++k) {
                    if(((input[(i * bit_lengthFcL)+k] & weights[((i * output_sizeFcL + j) * bit_lengthFcL)+k])) == 1)
                        {accumulatedOnes_pos++;}
                }
            } else if ((input[(i * bit_lengthFcL)]) * (weights[((i * output_sizeFcL + j) * bit_lengthFcL)]) == -1){
                for (size_t k = 1; k < bit_lengthFcL; ++k) {
                    if(((input[(i * bit_lengthFcL)+k] & weights[((i * output_sizeFcL + j) * bit_lengthFcL)+k])) == 1)
                        {accumulatedOnes_neg++;}
                }
            }
        }

        if (biases[(j * bit_lengthFcL)] == 1) {
            for (size_t biasIndex = 1; biasIndex < bit_lengthFcL; ++biasIndex) {
                accumulatedOnes_pos += biases[(j * bit_lengthFcL)+biasIndex];
            }
        } else if (biases[(j * bit_lengthFcL)] == -1) {
            for (size_t biasIndex = 1; biasIndex < bit_lengthFcL; ++biasIndex) {
                accumulatedOnes_neg += biases[(j * bit_lengthFcL)+biasIndex];
            }
        }
        numberOfAccumulations++;
        
        output[j] = accumulatedOnes_pos - accumulatedOnes_neg;
    }
}

std::vector<float> ScCudaFcLayer(
    const std::vector<float>& inputs, 
    const std::vector<float>& weights, 
    const std::vector<float>& biases, 
    const std::vector<float>& randomMatrix_input, 
    const std::vector<float>& randomMatrix_weights, 
    const std::vector<float>& randomMatrix_biases, 
    const int num_Outputs,
    const int bitstream_Length){

    // Create CUDA streams
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    
    // Determine the type of the input vector (2D or 3D)
    std::vector<float> RM_flat = randomMatrix_input;

    int inputData_size = inputs.size();
    int RM_cols = bitstream_Length;
    int output_size = inputData_size * RM_cols;

    float* d_inputData;
    float* d_RM;
    int8_t* d_output;

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (output_size + blockSize - 1) / blockSize;

    // Allocate input device memory
    cudaCheckError(hipMalloc(&d_inputData, inputData_size * sizeof(float)));
    cudaCheckError(hipMalloc(&d_RM, inputData_size * RM_cols * sizeof(float)));
    cudaCheckError(hipMalloc(&d_output, output_size * sizeof(int8_t)));

    ////STG__WEIGHTS__KERNEL//////
    std::vector<float> RM_flatW = randomMatrix_weights;

    std::vector<float> weights_data = weights;
    int inputData_sizeW = weights.size();
    int output_sizeW = inputData_sizeW * RM_cols;

    float* d_inputDataW;
    float* d_RMW;
    int8_t* d_outputW;

    // Define grid kernel and block kernel dimensions
    int blockSizeW = 256;
    int numBlocksW = (output_sizeW + blockSizeW - 1) / blockSizeW;      

    // Allocate kernel device memory
    cudaCheckError(hipMalloc(&d_inputDataW, inputData_sizeW * sizeof(float)));
    cudaCheckError(hipMalloc(&d_RMW, inputData_sizeW * RM_cols * sizeof(float)));
    cudaCheckError(hipMalloc(&d_outputW, output_sizeW * sizeof(int8_t)));

    ////STG__BIASES__KERNEL//////
    std::vector<float> RM_flatB = randomMatrix_biases;

    float* d_inputDataB;
    float* d_RMB;
    int8_t* d_outputB;

    int inputData_sizeB = biases.size();
    int output_sizeB = inputData_sizeB * RM_cols;

    // Define grid kernel and block kernel dimensions
    int blockSizeB = 256;
    int numBlocksB = (output_sizeB + blockSizeB - 1) / blockSizeB;      

    // Allocate kernel device memory
    cudaCheckError(hipMalloc(&d_inputDataB, inputData_sizeB * sizeof(float)));
    cudaCheckError(hipMalloc(&d_RMB, inputData_sizeB * RM_cols * sizeof(float)));
    cudaCheckError(hipMalloc(&d_outputB, output_sizeB * sizeof(int8_t)));

    //int output_sizeFc = biases.size();

    ///////OUTPUT SCFC__KERNEL//////
        // Number of output neurons
    int output_sizeFc =  num_Outputs;
        // Number of input neurons
    const int input_sizeFc = inputs.size(); 

    std::vector<float> h_output(output_sizeFc, 0);
    float* d_outputFc;
    cudaCheckError(hipMalloc(&d_outputFc, output_sizeFc * sizeof(float)));
    int blockSizeFc = 256;
    int numBlocksFc = (output_sizeFc + blockSizeFc - 1) / blockSizeFc;

                    //time measure
                    hipEvent_t start, stop;
                    hipEventCreate(&start);
                    hipEventCreate(&stop);
                    hipEventRecord(start);

    //CONVERTION INPUT AND KERNEL DATA TO STOCHASTIC
    // Copy INPUT data to device
    cudaCheckError(hipMemcpyAsync(d_inputData, inputs.data(), inputData_size * sizeof(float), hipMemcpyHostToDevice, stream1));
    cudaCheckError(hipMemcpyAsync(d_RM, RM_flat.data(), inputData_size * RM_cols * sizeof(float), hipMemcpyHostToDevice, stream1));

    // Copy WEIGHTS data to device
    cudaCheckError(hipMemcpyAsync(d_inputDataW, weights_data.data(), inputData_sizeW * sizeof(float), hipMemcpyHostToDevice, stream2));
    cudaCheckError(hipMemcpyAsync(d_RMW, RM_flatW.data(), inputData_sizeW * RM_cols * sizeof(float), hipMemcpyHostToDevice, stream2));

    // Copy BIAS data to device
    cudaCheckError(hipMemcpyAsync(d_inputDataB, biases.data(), inputData_sizeB * sizeof(float), hipMemcpyHostToDevice, stream3));
    cudaCheckError(hipMemcpyAsync(d_RMB, RM_flatB.data(), inputData_sizeB * RM_cols * sizeof(float), hipMemcpyHostToDevice, stream3));

    hipMemcpyToSymbol(HIP_SYMBOL(input_sizeFcL), &input_sizeFc, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(output_sizeFcL), &output_sizeFc, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bit_lengthFcL), &RM_cols, sizeof(int));

    ////STG__INPUT__KERNEL//////
    stochasticTensorGenerator<<<numBlocks, blockSize, 0, stream1>>>(d_inputData, d_RM, d_output, inputData_size, RM_cols);
    cudaCheckError(hipGetLastError());
    
    ////STG__WEIGHTS__KERNEL//////
    stochasticTensorGenerator<<<numBlocksW, blockSizeW, 0, stream2>>>(d_inputDataW, d_RMW, d_outputW, inputData_sizeW, RM_cols);
    cudaCheckError(hipGetLastError());

    ////STG__BIAS__KERNEL//////
    stochasticTensorGenerator<<<numBlocksB, blockSizeB, 0, stream3>>>(d_inputDataB, d_RMB, d_outputB, inputData_sizeB, RM_cols);
    cudaCheckError(hipGetLastError());

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);

    ////__SCFCLAYER__KERNEL//////
    forward_pass<<<numBlocksFc, blockSizeFc>>>(d_output, d_outputW, d_outputB, d_outputFc);
    cudaCheckError(hipGetLastError());

    //hipDeviceSynchronize();
    cudaCheckError(hipMemcpy(h_output.data(), d_outputFc, h_output.size() * sizeof(float), hipMemcpyDeviceToHost));

            //STOP time measure
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "\n" << std::endl;
            std::cout << "C++ ScFcLayer End-to-end execution time: " << milliseconds << " ms_" << std::endl;   

    // Free device memory
    hipFree(d_inputData);
    hipFree(d_RM);
    hipFree(d_output);

    hipFree(d_inputDataW);
    hipFree(d_RMW);
    hipFree(d_outputW);

    if (!biases.empty()) {
        hipFree(d_inputDataB);
        hipFree(d_RMB);
        hipFree(d_outputB);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_outputFc);

    return h_output;
}